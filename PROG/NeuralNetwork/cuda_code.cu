#include <hip/hip_runtime.h>


#define BLOCK_SIZE 16

typedef struct{
    int width;
    int height;
    int stride;

    float *elements;
} Matrix;

__global__
void matMulKernel(Matrix C, const Matrix A, const Matrix B){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = row*C.width + col;

    float out = 0;
    if(idx < C.width * C.height){
        for(int j=0; j < A.width; ++j)
            out += A.elements[row*A.width + j] * B.elements[j*B.width + col];
    }

    C.elements[idx] = out;

}


void matMul_wrapper(Matrix &C, const Matrix &A, const Matrix &B, hipDeviceProp_t devProp){
    dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 grid(  (C.width + block.x - 1) / block.x,
                (C.height + block.y - 1) / block.y,
                1);

    Matrix d_A {A.width, A.height, A.stride};
    size_t size = A.height * A.width * sizeof(float);
    hipMallocManaged(&d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);

    Matrix d_B {B.width, B.height, B.stride};
    size = B.height * B.width * sizeof(float);
    hipMallocManaged(&d_B.elements, size);
    hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);

    Matrix d_C {C.width, C.height, C.stride};
    size = C.height * C.width * sizeof(float);
    hipMallocManaged(&d_C.elements, size);
    hipMemcpy(d_C.elements, C.elements, size, hipMemcpyHostToDevice);

    matMulKernel<<<grid, block>>>(d_C, d_A, d_B);

    hipDeviceSynchronize();

    hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);

    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
}